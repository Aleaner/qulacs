#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime.h>
// for using cublas 
#include <hipblas.h>

#include <cstdio>
#include <cmath>
#include <cstdlib>
#include <cstring>
#include <complex>
#include <assert.h>
#include <algorithm>
#include <hip/hip_complex.h>
#include "util.h"
#include "util_common.h"
#include "util.cuh"

// return GTYPE*
GTYPE* allocate_quantum_state(ITYPE dim){
	GTYPE *state_gpu;
	checkCudaErrors(hipSetDevice(0));
	checkCudaErrors(hipMalloc((void**)&state_gpu, dim * sizeof(GTYPE)));
	//void* psi_gpu = reinterpret_cast<void*>(state_gpu);
    return state_gpu;
}

void initialize_quantum_state(GTYPE *state_gpu, ITYPE dim){
	//GTYPE* state_gpu = reinterpret_cast<hipDoubleComplex*>(psi_gpu);
	hipError_t cudaStatus;
	unsigned int block = dim <= 1024 ? dim : 1024;
	unsigned int grid = dim / block;
	init_qstate << <grid, block >> >(state_gpu, dim);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "init_state_gpu failed: %s\n", hipGetErrorString(cudaStatus));
	}
    //print_quantum_state(state_gpu, dim);
    //psi_gpu = reinterpret_cast<void*>(state_gpu);
}


void get_quantum_state(GTYPE* psi_gpu, void* psi_cpu_copy, ITYPE dim){
    // GTYPE* psi_gpu = reinterpret_cast<GTYPE*>(state_gpu);
    // CTYPE* state_cpu=(CTYPE*)malloc(sizeof(CTYPE)*dim);
    psi_cpu_copy = reinterpret_cast<CTYPE*>(psi_cpu_copy);
    checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipMemcpy(psi_cpu_copy, psi_gpu, dim * sizeof(CTYPE), hipMemcpyDeviceToHost));
    //state_cpu = reinterpret_cast<void*>(state_cpu);
    //print_quantum_state(psi_gpu, dim);
	//return psi_cpu_copy;
}

ITYPE insert_zero_to_basis_index(ITYPE basis_index, unsigned int qubit_index){
    ITYPE temp_basis = (basis_index >> qubit_index) << (qubit_index+1);
    return temp_basis + (basis_index & ( (1ULL<<qubit_index) -1));
}

void get_Pauli_masks_partial_list(const UINT* target_qubit_index_list, const UINT* Pauli_operator_type_list, UINT target_qubit_index_count, 
    ITYPE* bit_flip_mask, ITYPE* phase_flip_mask, UINT* global_phase_90rot_count, UINT* pivot_qubit_index){
    (*bit_flip_mask)=0;
    (*phase_flip_mask)=0;
    (*global_phase_90rot_count)=0;
    (*pivot_qubit_index)=0;
    for(UINT cursor=0;cursor < target_qubit_index_count; ++cursor){
        UINT target_qubit_index = target_qubit_index_list[cursor];
        switch(Pauli_operator_type_list[cursor]){
        case 0: // I
            break;
        case 1: // X
            (*bit_flip_mask) ^= 1ULL << target_qubit_index;
            (*pivot_qubit_index) = target_qubit_index;
            break;
        case 2: // Y
            (*bit_flip_mask) ^= 1ULL << target_qubit_index;
            (*phase_flip_mask) ^= 1ULL << target_qubit_index;
            (*global_phase_90rot_count) ++;
            (*pivot_qubit_index) = target_qubit_index;
            break;
        case 3: // Z
            (*phase_flip_mask) ^= 1ULL << target_qubit_index;
            break;
        default:
            fprintf(stderr,"Invalid Pauli operator ID called");
            assert(0);
        }
    }
}

void get_Pauli_masks_whole_list(const UINT* Pauli_operator_type_list, UINT target_qubit_index_count, 
    ITYPE* bit_flip_mask, ITYPE* phase_flip_mask, UINT* global_phase_90rot_count, UINT* pivot_qubit_index){

    (*bit_flip_mask)=0;
    (*phase_flip_mask)=0;
    (*global_phase_90rot_count)=0;
    (*pivot_qubit_index)=0;
    for(UINT target_qubit_index=0; target_qubit_index < target_qubit_index_count; ++target_qubit_index){
        switch(Pauli_operator_type_list[target_qubit_index]){
        case 0: // I
            break;
        case 1: // X
            (*bit_flip_mask) ^= 1ULL << target_qubit_index;
            (*pivot_qubit_index) = target_qubit_index;
            break;
        case 2: // Y
            (*bit_flip_mask) ^= 1ULL << target_qubit_index;
            (*phase_flip_mask) ^= 1ULL << target_qubit_index;
            (*global_phase_90rot_count) ++;
            (*pivot_qubit_index) = target_qubit_index;
            break;
        case 3: // Z
            (*phase_flip_mask) ^= 1ULL << target_qubit_index;
            break;
        default:
            fprintf(stderr,"Invalid Pauli operator ID called");
            assert(0);
        }
    }
}

ITYPE* create_matrix_mask_list(const UINT* qubit_index_list, UINT qubit_index_count){
    const ITYPE matrix_dim = 1ULL << qubit_index_count;
    ITYPE* mask_list = (ITYPE*) calloc((size_t)matrix_dim, sizeof(ITYPE));
    ITYPE cursor = 0;

    for(cursor=0;cursor < matrix_dim; ++cursor){
        for(UINT bit_cursor = 0; bit_cursor < qubit_index_count;++bit_cursor){
			if ((cursor >> bit_cursor) & 1) {
				UINT bit_index = qubit_index_list[bit_cursor];
				mask_list[cursor] ^= (1ULL << bit_index);
			}
        }
    }
    return mask_list;
}

UINT* create_sorted_ui_list(const UINT* array, size_t size){
    UINT* new_array = (UINT*)calloc(size,sizeof(UINT));
    memcpy(new_array, array, size*sizeof(UINT));
    std::sort(new_array, new_array+size);
    return new_array;
}

// C=alpha*A*B+beta*C
// in this wrapper, we assume beta is always zero!
int cublas_zgemm_wrapper(ITYPE n, CTYPE alpha, const CTYPE *h_A, const CTYPE *h_B, CTYPE beta, CTYPE *h_C){
    ITYPE n2 = n*n;
    hipblasStatus_t status;
    hipblasHandle_t handle;
    GTYPE *d_A;// = make_hipDoubleComplex(0.0,0.0);
    GTYPE *d_B;// = make_hipDoubleComplex(0,0);
    GTYPE *d_C;// = make_hipDoubleComplex(0,0);
    GTYPE d_alpha=make_hipDoubleComplex(alpha.real(), alpha.imag());
    GTYPE d_beta=make_hipDoubleComplex(beta.real(), beta.imag());
    int dev = 0; //findCudaDevice(argc, (const char **)argv);
    
    /* Initialize CUBLAS */
    status = hipblasCreate(&handle);

    if (status != HIPBLAS_STATUS_SUCCESS){
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return EXIT_FAILURE;
    }

    /* Allocate device memory for the matrices */
    if (hipMalloc(reinterpret_cast<void **>(&d_A), n2 * sizeof(d_A[0])) != hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate A)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc(reinterpret_cast<void **>(&d_B), n2 * sizeof(d_B[0])) != hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate B)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc(reinterpret_cast<void **>(&d_C), n2 * sizeof(d_C[0])) != hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate C)\n");
        return EXIT_FAILURE;
    }

    /* Initialize the device matrices with the host matrices */
    //status = hipblasSetVector(n2, sizeof(h_A[0]), h_A, 1, d_A, 1);
    status = hipblasSetMatrix(n, n, sizeof(h_A[0]), h_A, n, d_A, n);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (write A)\n");
        return EXIT_FAILURE;
    }

    //status = hipblasSetVector(n2, sizeof(h_B[0]), h_B, 1, d_B, 1);
    status = hipblasSetMatrix(n, n, sizeof(h_B[0]), h_B, n, d_B, n);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (write B)\n");
        return EXIT_FAILURE;
    }

    //status = hipblasSetVector(n2, sizeof(h_C[0]), h_C, 1, d_C, 1);
    status = hipblasSetMatrix(n, n, sizeof(h_C[0]), h_C, n, d_C, n);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (write C)\n");
        return EXIT_FAILURE;
    }
    /* Performs operation using cublas */
    status = hipblasZgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n, &d_alpha, d_A,
        n, d_B, n, &d_beta, d_C, n);

    //status=hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! kernel execution error.\n");
        return EXIT_FAILURE;
    }

    /* Allocate host memory for reading back the result from device memory */
    CTYPE* tmp_h_C = reinterpret_cast<CTYPE *>(malloc(n2 * sizeof(h_C[0])));

    if (tmp_h_C == 0) {
        fprintf(stderr, "!!!! host memory allocation error (C)\n");
        return EXIT_FAILURE;
    }

    /* Read the result back */
    status = hipblasGetMatrix(n, n, sizeof(GTYPE), d_C, n, tmp_h_C, n);
    memcpy(h_C, tmp_h_C, sizeof(h_C[0])*n2);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (read C)\n");
        return EXIT_FAILURE;
    }
    if (hipFree(d_A) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (A)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(d_B) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (B)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(d_C) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (C)\n");
        return EXIT_FAILURE;
    }

    /* Shutdown */
    status = hipblasDestroy(handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! shutdown error (A)\n");
        return EXIT_FAILURE;
    }
    return 0;
}

// C=alpha*A*x+beta*y
// in this wrapper, we assume beta is always zero!
int cublas_zgemv_wrapper(ITYPE n, CTYPE alpha, const CTYPE *h_A, const CTYPE *h_x, CTYPE beta, CTYPE *h_y){
    ITYPE n2 = n*n;
    hipblasStatus_t status;
    hipblasHandle_t handle;
    GTYPE *d_A;
    GTYPE *d_x;
    GTYPE *d_y;
    GTYPE d_alpha=make_hipDoubleComplex(alpha.real(), alpha.imag());
    GTYPE d_beta=make_hipDoubleComplex(beta.real(), beta.imag());
    int dev = 0; //findCudaDevice(argc, (const char **)argv);
    
    /* Initialize CUBLAS */
    printf("simpleCUBLAS test running..\n");
    status = hipblasCreate(&handle);

    if (status != HIPBLAS_STATUS_SUCCESS){
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return EXIT_FAILURE;
    }

    /* Allocate device memory for the matrices */
    if (hipMalloc(reinterpret_cast<void **>(&d_A), n2 * sizeof(d_A[0])) != hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate A)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc(reinterpret_cast<void **>(&d_x), n * sizeof(d_x[0])) != hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate x)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc(reinterpret_cast<void **>(&d_y), n * sizeof(d_y[0])) != hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate y)\n");
        return EXIT_FAILURE;
    }

    /* Initialize the device matrices with the host matrices */
    //status = hipblasSetVector(n2, sizeof(h_A[0]), h_A, 1, d_A, 1);
    status = hipblasSetMatrix(n, n, sizeof(h_A[0]), h_A, n, d_A, n);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (write A)\n");
        return EXIT_FAILURE;
    }

    status = hipblasSetVector(n, sizeof(h_x[0]), h_x, 1, d_x, 1);
    //status = hipblasSetMatrix(n, n, sizeof(h_B[0]), h_B, n, d_B, n);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (write x)\n");
        return EXIT_FAILURE;
    }

    status = hipblasSetVector(n, sizeof(h_y[0]), h_y, 1, d_y, 1);
    //status = hipblasSetMatrix(n, n, sizeof(h_C[0]), h_C, n, d_C, n);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (write C)\n");
        return EXIT_FAILURE;
    }
    /* Performs operation using cublas */
    status = hipblasZgemv(handle, HIPBLAS_OP_T, n, n, &d_alpha, d_A, n,
        d_x, 1, &d_beta, d_y, 1);
/*
hipblasStatus_t hipblasZgemv(hipblasHandle_t handle, hipblasOperation_t trans,
                           int m, int n,
                           const hipDoubleComplex *alpha,
                           const hipDoubleComplex *A, int lda,
                           const hipDoubleComplex *x, int incx,
                           const hipDoubleComplex *beta,
                           hipDoubleComplex *y, int incy)
*/
    //status=hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! kernel execution error.\n");
        return EXIT_FAILURE;
    }

    /* Allocate host memory for reading back the result from device memory */
    CTYPE* tmp_h_y = reinterpret_cast<CTYPE *>(malloc(n * sizeof(h_y[0])));

    if (tmp_h_y == 0) {
        fprintf(stderr, "!!!! host memory allocation error (y)\n");
        return EXIT_FAILURE;
    }

    /* Read the result back */
    status = hipblasGetVector(n, sizeof(GTYPE), d_y, 1, tmp_h_y, 1);
    /*
    hipblasStatus_t hipblasGetVector(int n, int elemSize, const void *x, int incx, void *y, int incy)
    */
    memcpy(h_y, tmp_h_y, sizeof(h_y[0])*n);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (read C)\n");
        return EXIT_FAILURE;
    }
    if (hipFree(d_A) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (A)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(d_x) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (x)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(d_y) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (y)\n");
        return EXIT_FAILURE;
    }

    /* Shutdown */
    status = hipblasDestroy(handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! shutdown error (A)\n");
        return EXIT_FAILURE;
    }
    return 0;
}

// we assume state has already allocated at device
int cublas_zgemv_wrapper(ITYPE n, const CTYPE *h_matrix, GTYPE *d_state){
    ITYPE n2 = n*n;
    hipblasStatus_t status;
    hipblasHandle_t handle;
    GTYPE *d_matrix;
    GTYPE *d_y; // this will include the answer of the state.
    GTYPE d_alpha = make_hipDoubleComplex(1.0, 0.0);
    GTYPE d_beta = make_hipDoubleComplex(0.0, 0.0);
    int dev = 0;
    
    /* Initialize CUBLAS */
    status = hipblasCreate(&handle);

    if (status != HIPBLAS_STATUS_SUCCESS){
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return EXIT_FAILURE;
    }

    /* Allocate device memory for the matrices */
    if (hipMalloc(reinterpret_cast<void **>(&d_matrix), n2 * sizeof(d_matrix[0])) != hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate A)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc(reinterpret_cast<void **>(&d_y), n * sizeof(d_y[0])) != hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate y)\n");
        return EXIT_FAILURE;
    }
    // hipMemset(&d_y, 0, sizeof(d_y[0])*n);
    /* Initialize the device matrices with the host matrices */
    status = hipblasSetMatrix(n, n, sizeof(h_matrix[0]), h_matrix, n, d_matrix, n);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (write A)\n");
        return EXIT_FAILURE;
    }

    /* Performs operation using cublas */
    status = hipblasZgemv(handle, HIPBLAS_OP_T, n, n, &d_alpha, d_matrix, n,
        d_state, 1, &d_beta, d_y, 1);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! kernel execution error.\n");
        return EXIT_FAILURE;
    }

    hipMemcpy(d_state, d_y, n * sizeof(GTYPE), hipMemcpyDeviceToDevice);
   
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (read C)\n");
        return EXIT_FAILURE;
    }
    if (hipFree(d_matrix) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (A)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(d_y) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (y)\n");
        return EXIT_FAILURE;
    }

    /* Shutdown */
    status = hipblasDestroy(handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! shutdown error (A)\n");
        return EXIT_FAILURE;
    }
    return 0;
}
